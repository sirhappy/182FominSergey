
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>


__global__ void add_OneBlockOneThread(int n, float *x, float *y, float *z)
{
	for (int i = 0; i < n; i++)
		z[i] = x[i] + y[i];
}


__global__ void add_OneBlockManyThreads(int n, float *x, float *y, float *z)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
		z[i] = x[i] + y[i];
}

__global__ void add_ManyBlocksManyThreads(int n, float *x, float *y, float *z)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		z[i] = x[i] + y[i];
}

int main(void)
{

	int N = 1 << 23; // 1048576 elements
	float *x, *y, *z;
	std::chrono::steady_clock::time_point start, end;

	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));
	hipMallocManaged(&z, N * sizeof(float));

	// ========== One block, one thread ========== //
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	start = std::chrono::steady_clock::now();

	add_OneBlockOneThread <<< 1, 1 >>> (N, x, y, z);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms" << std::endl;


	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	// ========== One block, many threads ========== //
	start = std::chrono::steady_clock::now();

	add_OneBlockManyThreads <<< 1, 512 >>> (N, x, y, z);

	hipDeviceSynchronize();
	end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms" << std::endl;

	
	// ========== Many Blocks, many threads ========== //
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	
	start = std::chrono::steady_clock::now();

	int blockSize = 512;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add_ManyBlocksManyThreads <<< numBlocks, blockSize >>> (N, x, y, z);

	hipDeviceSynchronize();
	end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms" << std::endl;


	// Free memory
	hipFree(x);
	hipFree(y);
	hipFree(z);

	return 0;
}
