
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>


__global__ void add_OneBlockOneThread(int n, float *x, float *y, float *z)
{
	for (int i = 0; i < n; i++)
		z[i] = x[i] + y[i];
}


__global__ void add_OneBlockManyThreads(int n, float *x, float *y, float *z)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
		z[i] = x[i] + y[i];
}

__global__ void add_ManyBlocksManyThreads(int n, float *x, float *y, float *z)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		z[i] = x[i] + y[i];
}

__host__ void add_SingleThreadCPU(int n, float *x, float *y, float *z)
{
	for (int i = 0; i < n; ++i)
		z[i] = x[i] + y[i];
}

int main(void)
{

	int N = 1 << 20; // 1048576 elements
	float *x, *y, *z;

	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));
	hipMallocManaged(&z, N * sizeof(float));

	// ========== CPU, one thread ========== //
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	add_SingleThreadCPU(N, x, y, z);
	hipDeviceSynchronize();

	// ========== One block, one thread ========== //
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	add_OneBlockOneThread <<< 1, 1 >>> (N, x, y, z);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();


	// ========== One block, many threads ========== //
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	add_OneBlockManyThreads <<< 1, 512 >>> (N, x, y, z);
	hipDeviceSynchronize();

	// ========== Many Blocks, many threads ========== //
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	int blockSize = 512;
	int numBlocks = (N + blockSize - 1) / blockSize;

	add_ManyBlocksManyThreads <<< numBlocks, blockSize >>> (N, x, y, z);
	hipDeviceSynchronize();



	hipFree(x);
	hipFree(y);
	hipFree(z);

	return 0;
}
