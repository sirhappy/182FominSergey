
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>


__global__ void add_OneBlockOneThread(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++)
		y[i] = x[i] + y[i];
}


__global__ void add_OneBlockManyThreads(int n, float *x, float *y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

__global__ void add_ManyBlocksManyThreads(int n, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main(void)
{

	int N = 1 << 20; // 1048576 elements
	float *x, *y;
	std::chrono::steady_clock::time_point start, end;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));

	int numberOfThreads;
	std::cout << "Input number of threads: ";
	std::cin >> numberOfThreads;
	std::cout << std::endl;

	// ========== One block, one thread ========== //
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	start = std::chrono::steady_clock::now();

	add_OneBlockOneThread <<< 1, 1 >>> (N, x, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms" << std::endl;


	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	// ========== One block, many threads ========== //
	start = std::chrono::steady_clock::now();

	add_OneBlockManyThreads <<< 1, numberOfThreads >>> (N, x, y);

	hipDeviceSynchronize();
	end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms" << std::endl;

	
	// ========== Many Blocks, many threads ========== //
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	
	start = std::chrono::steady_clock::now();

	int blockSize = numberOfThreads;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add_ManyBlocksManyThreads <<< numBlocks, blockSize >>> (N, x, y);

	hipDeviceSynchronize();
	end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms" << std::endl;


	// Free memory
	hipFree(x);
	hipFree(y);
	
	return 0;
}
