
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <iostream>

__global__ void transposeMatrix(const int *a, const int *b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
}

int main()
{
	const int MATRIX_SIZE = 3;
	const int N = MATRIX_SIZE * MATRIX_SIZE;

	float *a, *b, *c;
	hipMallocManaged(&a, N * sizeof(float));
	hipMallocManaged(&b, N * sizeof(float));

	for (int i = 0; i < N; ++i)
	{
		a[i] = 1.0f;
		b[i] = 2.0f;
	}

	
    return 0;
}
